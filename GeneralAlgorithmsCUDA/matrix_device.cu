#include <hip/hip_runtime.h>
#include "matrix_device.h"
#include <utility>

namespace CudaPlayground
{
    CUDA_HOST_API DeviceMatrixGuard<mat_fr> toDeviceMemory(const mat_fr host, bool copy)
    {
        size_t stride;
        float* data;
        checkCudaErrors(hipMallocPitch(&data, &stride, host.cols * sizeof(float), host.rows));
        if (copy)
        {
            checkCudaErrors(hipMemcpy2D(
                data, stride,
                host.elements, host.stride * sizeof(float),
                host.cols * sizeof(float), host.rows,
                hipMemcpyKind::hipMemcpyHostToDevice
            ));
        }

        return mat_fr{ host.rows, host.cols, (int)(stride/sizeof(float)), data };
    }

    CUDA_HOST_API DeviceMatrixGuard<mat_fc> toDeviceMemory(const mat_fc host, bool copy)
    {
        size_t stride;
        float* data;
        checkCudaErrors(hipMallocPitch(&data, &stride, host.rows * sizeof(float), host.cols));
        if (copy)
        {
            checkCudaErrors(hipMemcpy2D(
                data, stride,
                host.elements, host.stride * sizeof(float),
                host.rows * sizeof(float), host.cols,
                hipMemcpyKind::hipMemcpyHostToDevice
            ));
        }

        return mat_fc{ host.rows, host.cols, (int)(stride / sizeof(float)), data };
    }

    CUDA_HOST_API DeviceMatrixGuard<mat_fr> toDeviceMemoryPad(
        const mat_fr host,
        int blockSizeRows,
        int blockSizeCols,
        bool copy)
    {
        int rows = pad(host.rows, blockSizeRows);
        int cols = pad(host.cols, blockSizeCols);

        size_t stride;
        float* data;
        checkCudaErrors(hipMallocPitch(&data, &stride, cols * sizeof(float), rows));
        checkCudaErrors(hipMemset2D(data, stride, 0, cols * sizeof(float), rows));

        if (copy)
        {
            checkCudaErrors(hipMemcpy2D(
                data, stride,
                host.elements, host.stride * sizeof(float),
                host.cols * sizeof(float), host.rows,
                hipMemcpyKind::hipMemcpyHostToDevice
            ));
        }
        return mat_fr{ rows, cols, (int)(stride / sizeof(float)), data };
    }

    CUDA_HOST_API DeviceMatrixGuard<mat_fr> toDeviceMemoryExtendedBlock(
        const mat_fr host,
        const mat_fr block,
        int blockSizeRows,
        int blockSizeCols,
        bool copy)
    {
        int extRows = host.rows + (block.rows-1)/2;
        int extCols = host.cols + (block.cols-1)/2;

        size_t stride;
        float* data;
        checkCudaErrors(hipMallocPitch(&data, &stride, extCols * sizeof(float), extRows));
        checkCudaErrors(hipMemset2D(data, stride, 0, extCols * sizeof(float), extRows));

        if (copy)
        {
            int dataStart = (block.rows / 2) * (stride / sizeof(float)) + block.cols / 2;
            checkCudaErrors(hipMemcpy2D(
                data + dataStart, (stride + (block.cols / 2)) * sizeof(float), // extra stride is for extended block
                host.elements, host.stride * sizeof(float),
                host.cols * sizeof(float), host.rows,
                hipMemcpyKind::hipMemcpyHostToDevice
            ));
        }
        return mat_fr{ extRows, extCols, (int)(stride / sizeof(float)), data };
    }

    inline CUDA_HOST_API void copyMemory(const mat_fr from, mat_fr& to, hipMemcpyKind kind)
    {
        checkCudaErrors(hipMemcpy2D(
            to.elements, to.stride * sizeof(float),
            from.elements, from.stride * sizeof(float),
            to.cols * sizeof(float), to.rows,
            kind
        ));
    }

    CUDA_HOST_API void copyFromDeviceMemory(const mat_fr dev, mat_fr& host)
    {
        copyMemory(dev, host, hipMemcpyKind::hipMemcpyDeviceToHost);
    }

    CUDA_HOST_API void copyFromHostMemory(const mat_fr host, mat_fr& dev)
    {
        copyMemory(host, dev, hipMemcpyKind::hipMemcpyHostToDevice);
    }

    CUDA_HOST_API void copyFromDeviceMemoryExtendedBlockPad(const mat_fr dev, const mat_fr block, mat_fr& host)
    {
        int dataStart = (block.rows / 2) * dev.stride + block.cols / 2;
        checkCudaErrors(hipMemcpy2D(
            host.elements, host.stride * sizeof(float),
            dev.elements + dataStart, (dev.stride + (block.cols / 2)) * sizeof(float),
            host.cols * sizeof(float), host.rows,
            hipMemcpyKind::hipMemcpyDeviceToHost
        ));
    }

    inline CUDA_HOST_API void copyMemory(const mat_fc from, mat_fc& to, hipMemcpyKind kind)
    {
        checkCudaErrors(hipMemcpy2D(
            to.elements, to.stride * sizeof(float),
            from.elements, from.stride * sizeof(float),
            to.rows * sizeof(float), to.cols,
            kind
        ));
    }

    CUDA_HOST_API void copyFromDeviceMemory(const mat_fc dev, mat_fc& host)
    {
        copyMemory(dev, host, hipMemcpyKind::hipMemcpyDeviceToHost);
    }
}

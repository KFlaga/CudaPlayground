#include "hip_stream.h"
#include "cuda_all.h"

namespace CudaPlayground
{
	CUDA_HOST_API CudaStream::CudaStream()
	{
		hipStreamCreate(&stream);
	}

	CUDA_HOST_API CudaStream::~CudaStream()
	{
		if (stream != nullptr)
		{
			hipStreamDestroy(stream);
		}
	}

	CUDA_HOST_API CudaStream::CudaStream(CudaStream&& other) noexcept
	{
		std::swap(stream, other.stream);
	}

	CUDA_HOST_API CudaStream& CudaStream::operator=(CudaStream&& other) noexcept
	{
		if (stream != nullptr)
		{
			hipStreamDestroy(stream);
			stream = nullptr;
		}
		std::swap(stream, other.stream);
		return *this;
	}

	CUDA_HOST_API bool CudaStream::isWorking()
	{
		return hipStreamQuery(stream) == ::hipErrorNotReady;
	}

	CUDA_HOST_API void CudaStream::sync()
	{
		if (isWorking())
		{
			hipStreamSynchronize(stream);
		}
	}
}

